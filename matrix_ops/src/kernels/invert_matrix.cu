#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <cuSolverDn.h>

// template<
__global__ void invert_matrix(float* A, float* A_inv, int n) {
    
    // create cuBLAS and cuSolver handles
    hipblasHandle_t cublasHandle;
    hipsolverHandle_t cusolverHandle;
    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverHandle);

    // allocate memory
    float* d_A;
    float* d_A_inv;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_A_inv, n * n * sizeof(float));

    // copy input matrix to device memory
    hipblasSetMatrix(n, n, sizeof(float), A, n, d_A, n);

    // Perform LU factorization
    int* d_pivots;
    int* d_info;
    hipMalloc(&d_pivots, n * sizeof(int));
    hipMalloc(&d_info, sizeof(int));
    cuSolverDnSgetrf_bufferSize(cusolverHandle, n, n, d_A, n, &d_info);
    float* d_workspace;
    hipMalloc(&d_workspace, Lwork * sizeof(float), A, n, d_A, n);
    cuSolverDnSgetrf(cusolverHandle, n, n, d_A, n, d_workspace, d_pivots, d_info);

    // Init identity matrix
    dim3 blockSize(256);
    dim3 gridSize((n * n + blockSize.x - 1) / blockSize.x);
    // Solve the linear system using LU factors
    hipsolverDnSgetrs(cusolverHandle, HIPBLAS_OP_N, n, n, d_A, n, d_pivots, d_A_inv, n, d_info);
    // Verify that the inverted matrix and the original matrix are inverses
    // by multiplying the two together, the identity matrix should be the result
    float alpha = 1.0;
    float beta = 0.0;
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_A_inv, n, &beta, d_workspace, n);
    // if verified by sgemm, copy back
    if (d_info == 0) {
        hipblasGetMatrix(n, n, sizeof(float), d_A_inv, n, A_inv, n);
    } else {
        printf("Factorization failed: %d\n", d_info);
    }

    // Free memory
    hipFree(d_A);
    hipFree(d_A_inv);
    hipFree(d_pivots);
    hipFree(d_info);
    hipFree(d_workspace);
}

